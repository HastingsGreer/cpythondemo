#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void mandel(double* ref_real_array,
                     double* ref_imag_array, 
                     double* dc_real_array, 
                     double* dc_imag_array, 
                     int depth,
                     int* count_array)
{
  unsigned int i = threadIdx.x + 512 * blockIdx.x;


  double dc_real = dc_real_array[i];
  double dc_imag = dc_imag_array[i];

  int count = 0;
  double d_real = 0;
  double d_imag = 0;
  double d_real_temp;
  while((d_real + ref_real_array[count]) * (d_real + ref_real_array[count]) + 
                   (d_imag + ref_imag_array[count]) * (d_imag + ref_imag_array[count]) < 4 && 
                   count < depth){
    
    double z_real = ref_real_array[count];
    double z_imag = ref_imag_array[count];
    d_real_temp = 2 * z_real * d_real - 2 * z_imag * d_imag + d_real * d_real - d_imag * d_imag + dc_real;
    d_imag = 2 * z_real * d_imag + 2 * z_imag * d_real + 2 * d_real * d_imag + dc_imag;
    d_real = d_real_temp;
    count ++;
  }
  count_array[i] = count;
  
}
extern "C" int cu_mandel(double* ref_real_array,
                     double* ref_imag_array, 
                     double* dc_real_array, 
                     double* dc_imag_array, 
                     int depth,
                     int* count_array, 
                     int l_ref){
    printf("entering function");

    double *dev_real_ref, *dev_imag_ref, *dev_dc_real, *dev_dc_imag;
    int *dev_counts;
    hipMalloc((void**)&dev_real_ref, l_ref * sizeof(double));
    hipMalloc((void**)&dev_imag_ref, l_ref * sizeof(double));
    hipMalloc((void**)&dev_dc_real, 512 * 512 *sizeof(double));
    hipMalloc((void**)&dev_dc_imag, 512 * 512 *sizeof(double));
    hipMalloc((void**)&dev_counts, 512 * 512 *sizeof(int));

    hipMemcpy(dev_real_ref, ref_real_array, l_ref * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_imag_ref, ref_imag_array, l_ref * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_dc_real, dc_real_array, 512 * 512 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_dc_imag, dc_imag_array, 512 * 512 * sizeof(double), hipMemcpyHostToDevice);
    printf("calling kernel");

    mandel<<<512, 512>>>(dev_real_ref, dev_imag_ref, dev_dc_real, dev_dc_imag, depth, dev_counts);


    hipMemcpy(count_array, dev_counts, 512 * 512 * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("a count: %d\n", count_array[0]);
    hipFree(dev_real_ref);
    hipFree(dev_imag_ref);
    hipFree(dev_dc_real);
    hipFree(dev_dc_imag);
    hipFree(dev_counts);
    return 0;

}